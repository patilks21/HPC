#include<iostream>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<time.h>

#define BLOCK_SIZE 64
#define SOA 512

void random_ints(int *data,int size)
{
	int i;
	for(i=0;i<size;i++)
	{
		data[i]=rand()%size;
	}
}

__global__ void ReductionMax2(int *input,int *results,int n)
{
	__shared__ int sdata[BLOCK_SIZE];
	unsigned int i=blockIdx.x*blockDim.x+threadIdx.x;
	unsigned int tx=threadIdx.x;
	int x=-INT_MAX;
	if(i<n)
		x=input[i];
	sdata[tx]=x;
	__syncthreads();
	for(unsigned int offset=blockDim.x>>1;offset>0;offset >>=1)
	{
		__syncthreads();
		if(tx<offset)
		{
			if(sdata[tx+offset]>sdata[tx])
				sdata[tx]=sdata[tx+offset];
		}
	}
	if(threadIdx.x==0)
	{
		results[blockIdx.x]=sdata[0];
	}
}
int  main()
{
	int num_blocks=SOA/BLOCK_SIZE;
	int num_threads=BLOCK_SIZE,i;
	unsigned int mem_size_a=sizeof(int)*SOA;
	int *h_a=(int*)malloc(mem_size_a);
	random_ints(h_a,SOA);
	int *d_a;
	hipMalloc((void**)&d_a,mem_size_a);
	hipMemcpy(d_a,h_a,mem_size_a,hipMemcpyHostToDevice);
	unsigned int mem_size_b=sizeof(int)*num_blocks;
	int *d_b;
	hipMalloc((void**)&d_b,mem_size_b);
	int *h_b=(int*)malloc(mem_size_b);
	unsigned int mem_size_c=sizeof(int);
	int *d_c;
	hipMalloc((void**)&d_c,mem_size_c);

	ReductionMax2<<<num_blocks,num_threads>>>(d_a,d_b,SOA);
	hipMemcpy(h_b,d_b,mem_size_b,hipMemcpyDeviceToHost);
	ReductionMax2<<<1,num_blocks>>>(d_b,d_c,num_blocks);

	int *h_c=(int*)malloc(mem_size_c);
	hipMemcpy(h_c,d_c,mem_size_c,hipMemcpyDeviceToHost);

	int j;
			for(j=0;j<SOA;j++)
			{
				std::cout<<h_a[j]<<",";
			}
			std::cout<<"\nblock max";
			for(j=0;j<num_blocks;j++)
			{
				std::cout<<h_b[j]<<",";
			}
			std::cout<<"\nparallel max="<<*h_c;
}
