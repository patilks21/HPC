
#include <hip/hip_runtime.h>
#include<iostream>
#include<stdlib.h>
__global__ void add(int *a,int *b,int *c)
{
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	c[index]=a[index]+b[index];
}
void random_ints(int *a,int N)
{
	int i;
	for(i=0;i<N;i++)
	{
		a[i]=i;
	}
}
#define N 2048
#define THREADS_PER_BLOCK 64

int main(void)
{
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;
	int size=N*sizeof(int);

	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	a=(int *)malloc(size);random_ints(a,N);
	b=(int *)malloc(size);random_ints(b,N);
	c=(int *)malloc(size);

	hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

	add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a,d_b,d_c);

	hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	int i;
		for(i=0;i<N;i++)
		{
			std::cout<<c[i]<<"\n";
		}

	return 0;
}
